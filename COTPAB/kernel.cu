#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2\opencv.hpp>
//#include <opencv2\cudaimgproc.hpp>

#include "slic.h"

using namespace std;
using namespace cv;

#define nc 80 //maximum vizsg�lt t�vols�g a centroidok keres�sekor
#define numberofSuperpixels 5000
#define iteration 10


int cols;
int rows;
int step;
int centersLength;
int *clusters;
float *distances;
float *centers;
int *center_counts;
uchar3 *colors;

__device__ int *d_clusters;			//1D --> cols * rows
__device__ float *d_distances;		//1D --> cols * rows
__device__ float *d_centers;		//1D --> centersLength * 5
__device__ int *d_center_counts;	//1D --> centersLength
__device__ uchar3 *d_colors;		//1D --> cols * rows

__device__ float compute_dist(int ci, int y, int x, uchar3 colour, float *d_centers, int pitch, int d_step)
{
	//sz�nt�vols�g
	float dc = sqrt(pow(d_centers[ci *pitch + 0] - colour.x, 2) + pow(d_centers[ci *pitch + 1] - colour.y, 2)
		+ pow(d_centers[ci *pitch + 2] - colour.z, 2));
	//euklideszi t�vols�g
	float ds = sqrt(pow(d_centers[ci *pitch + 3] - x, 2) + pow(d_centers[ci *pitch + 4] - y, 2));

	return sqrt(pow(dc / nc, 2) + pow(ds / d_step, 2));
}

__device__ void compute0(int clusterIDX, int d_cols, int d_rows, int d_step, int d_centersLength, int *d_clusters, float *d_distances,
	float *d_centers, int *d_center_counts, uchar3 *d_colors, int pitch)
{
	for (int pixelY = d_centers[clusterIDX *pitch + 3] - (d_step*1.5); pixelY < d_centers[clusterIDX *pitch + 3] + (d_step*1.5); pixelY++)
	{
		for (int pixelX = d_centers[clusterIDX *pitch + 4] - (d_step*1.5); pixelX < d_centers[clusterIDX *pitch + 4] + (d_step*1.5); pixelX++)
		{

			if (pixelX >= 0 && pixelX < d_rows && pixelY >= 0 && pixelY < d_cols) {

				uchar3 colour = d_colors[d_cols*pixelX + pixelY];

				float distance = compute_dist(clusterIDX, pixelX, pixelY, colour, d_centers, pitch, d_step);
				if (distance < d_distances[d_cols*pixelX + pixelY])
				{
					d_distances[d_cols*pixelX + pixelY] = distance;
					d_clusters[d_cols*pixelX + pixelY] = clusterIDX;
				}
			}
		}
	}
	//a centroidok alaphelyzetbe �ll�t�sa
	d_centers[clusterIDX *pitch + 0] = 0;
	d_centers[clusterIDX *pitch + 1] = 0;
	d_centers[clusterIDX *pitch + 2] = 0;
	d_centers[clusterIDX *pitch + 3] = 0;
	d_centers[clusterIDX *pitch + 4] = 0;
	d_center_counts[clusterIDX] = 0;
}

__device__ void compute1(int idIn1D, int d_cols, float *d_centers, int *d_center_counts, uchar3 *d_colors, int pitch)
{
		d_distances[idIn1D] = FLT_MAX;
		
		/*int whichCluster = d_clusters[idIn1D];
		d_centers[whichCluster*pitch + 0] += d_colors[idIn1D].x;
		d_centers[whichCluster*pitch + 1] += d_colors[idIn1D].y;
		d_centers[whichCluster*pitch + 2] += d_colors[idIn1D].z;
		d_centers[whichCluster*pitch + 3] += idIn1D / d_cols;
		d_centers[whichCluster*pitch + 4] += idIn1D % d_cols;

		atomicAdd(&d_center_counts[whichCluster], 1);*/
}


__global__ void compute(int d_cols, int d_rows, int d_step, int d_centersLength, int *d_clusters, float *d_distances,
	float *d_centers, int *d_center_counts, uchar3 *d_colors, int pitch)
{
	int howManyPixels = d_cols*d_rows - 1;
	int idIn1D = blockIdx.x * blockDim.x + threadIdx.x;
	//ha a sz�l id-je nagyobb mint a pixelek sz�ma, akkor az egy cluster
	//sz�l, amely sz�lnak az indexe itt ker�l inicializ�l�sra
	int clusterIDX = idIn1D - howManyPixels;

	//for (int i = 0; i < iteration; i++)
	//{
		if (idIn1D > howManyPixels)
		{
			for (int pixelY = d_centers[clusterIDX *pitch + 3] - (d_step*1.5); pixelY < d_centers[clusterIDX *pitch + 3] + (d_step*1.5); pixelY++)
			{
				for (int pixelX = d_centers[clusterIDX *pitch + 4] - (d_step*1.5); pixelX < d_centers[clusterIDX *pitch + 4] + (d_step*1.5); pixelX++)
				{

					if (pixelX >= 0 && pixelX < d_rows && pixelY >= 0 && pixelY < d_cols) {

						uchar3 colour = d_colors[d_cols*pixelX + pixelY];

						float distance = compute_dist(clusterIDX, pixelX, pixelY, colour, d_centers, pitch, d_step);
						if (distance < d_distances[d_cols*pixelX + pixelY])
						{
							d_distances[d_cols*pixelX + pixelY] = distance;
							d_clusters[d_cols*pixelX + pixelY] = clusterIDX;
						}
					}
				}
			}
			//a centroidok alaphelyzetbe �ll�t�sa
			d_centers[clusterIDX *pitch + 0] = 0;
			d_centers[clusterIDX *pitch + 1] = 0;
			d_centers[clusterIDX *pitch + 2] = 0;
			d_centers[clusterIDX *pitch + 3] = 0;
			d_centers[clusterIDX *pitch + 4] = 0;
			d_center_counts[clusterIDX] = 0;
		}
		__syncthreads();

		//if (idIn1D <= howManyPixels)
		//{
		//	compute1(idIn1D, d_cols, d_centers, d_center_counts, d_colors,  pitch);
		//}
		//	d_distances[idIn1D] = FLT_MAX;
		//	//printf("%f", d_distances[idIn1D]);
		//	int whichCluster = d_clusters[idIn1D];
		//	/*d_centers[whichCluster*pitch + 0] += d_colors[idIn1D].x;
		//	d_centers[whichCluster*pitch + 1] += d_colors[idIn1D].y;
		//	d_centers[whichCluster*pitch + 2] += d_colors[idIn1D].z;
		//	d_centers[whichCluster*pitch + 3] += idIn1D / d_cols;
		//	d_centers[whichCluster*pitch + 4] += idIn1D % d_cols;*/

		//	//atomicAdd(&d_center_counts[whichCluster], 1);

		//	/*int c_id = clusters[j][k];

		//	if (c_id != -1) {
		//		Vec3b colour = image.at<Vec3b>(k, j);

		//		centers[c_id][0] += colour.val[0];
		//		centers[c_id][1] += colour.val[1];
		//		centers[c_id][2] += colour.val[2];
		//		centers[c_id][3] += j;
		//		centers[c_id][4] += k;

		//		center_counts[c_id] += 1;
		//	}*/
		//}
		//__syncthreads();

	//}

	//d_distances[threadIdx.x] = compute_dist(885, threadIdx.x % d_rows, threadIdx.x / d_rows, d_colors[threadIdx.x], d_centers, pitch, d_step);
}

__global__ void compute1(int d_cols, int d_rows, int d_step, int d_centersLength, int *d_clusters, float *d_distances,
	float *d_centers, int *d_center_counts, uchar3 *d_colors, int pitch)
{
	int howManyPixels = d_cols*d_rows - 1;
	int idIn1D = blockIdx.x * blockDim.x + threadIdx.x;
	//ha a sz�l id-je nagyobb mint a pixelek sz�ma, akkor az egy cluster
	//sz�l, amely sz�lnak az indexe itt ker�l inicializ�l�sra
	int clusterIDX = idIn1D - howManyPixels;

	if (idIn1D <= howManyPixels)
	{
		d_distances[idIn1D] = FLT_MAX;
		//printf("%f", d_distances[idIn1D]);
		int whichCluster = d_clusters[idIn1D];
		d_centers[whichCluster*pitch + 0] += d_colors[idIn1D].x;
		d_centers[whichCluster*pitch + 1] += d_colors[idIn1D].y;
		d_centers[whichCluster*pitch + 2] += d_colors[idIn1D].z;
		d_centers[whichCluster*pitch + 3] += idIn1D / d_cols;
		d_centers[whichCluster*pitch + 4] += idIn1D % d_cols;

		atomicAdd(&d_center_counts[whichCluster], 1);
	}
	__syncthreads();
}

void initData(Mat image)
{
	clusters = new int[cols*rows];
	distances = new float[cols*rows];
	for (int i = 0; i < cols*rows; i++)
	{
		clusters[i] = -1;
		distances[i] = FLT_MAX;
	}

	//Ez az�rt kell mert el�re nem tudom, hogy h�ny eleme lesz a centers-nek, ez�rt el�sz�r egy vectorhoz adomgatom hozz� az elemeket
	// majd k�s�bb l�trehozom a t�mb�t annyi elemmel, ah�ny eleme van a seg�dvectornak, majd �tm�solom az adatokat.
	vector<vector<float> > h_centers;
	for (int i = step; i < cols - step / 2; i += step) {
		for (int j = step; j < rows - step / 2; j += step) {
			vector<float> center;
			/* Find the local minimum (gradient-wise). */
			//Point nc = find_local_minimum(image, Point(i, j));
			Vec3b colour = image.at<Vec3b>(j, i);//nc.y, nc.x);

			center.push_back(colour.val[0]);
			center.push_back(colour.val[1]);
			center.push_back(colour.val[2]);
			center.push_back(i);//nc.x);
			center.push_back(j);//nc.y);

			h_centers.push_back(center);
		}
	}

	centersLength = h_centers.size();

	centers = new float[centersLength * 5];
	center_counts = new int[centersLength];
	int idx = 0;
	for (int i = 0; i < centersLength; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			centers[idx] = h_centers[i][j];
			idx++;
		}
		center_counts[i] = 0;
	}

	//Bej�rom a k�pet, majd minden pixel sz�n�t (3 �rt�k) elmentem egy uchar3 v�ltoz�ba
	colors = new uchar3[rows*cols];
	for (int i = 0; i < cols; i++)
	{
		for (int j = 0; j < rows; j++)
		{
			Vec3b colour = image.at<Vec3b>(j, i);
			colors[i * rows + j] = make_uchar3(colour.val[0], colour.val[1], colour.val[2]);
		}
	}
}

void dataCopy()
{
	hipMalloc((void**)&d_clusters, sizeof(int)*rows*cols);
	hipMemcpy(d_clusters, clusters, sizeof(int)*rows*cols, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_distances, sizeof(float)*rows*cols);
	hipMemcpy(d_distances, distances, sizeof(float)*rows*cols, hipMemcpyHostToDevice);

	//size_t pitch = 5;
	//hipMallocPitch((void**)&d_centers, &pitch, sizeof(float) * centersLength, 5);
	//hipMemcpy2D(d_centers, pitch, centers, sizeof(float) * centersLength, sizeof(float) * centersLength, 5, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_centers, sizeof(float)*centersLength * 5);
	hipMemcpy(d_centers, centers, sizeof(float)*centersLength * 5, hipMemcpyHostToDevice);


	hipMalloc((void**)&d_center_counts, sizeof(int)*centersLength);
	hipMemcpy(d_center_counts, center_counts, sizeof(int)*centersLength, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_colors, sizeof(uchar3)*rows*cols);
	hipMemcpy(d_colors, colors, sizeof(uchar3)*rows*cols, hipMemcpyHostToDevice);
}

void dataFree()
{
	hipFree(d_clusters);
	hipFree(d_distances);
	hipFree(d_centers);
	hipFree(d_center_counts);
	hipFree(d_colors);
}

int main()
{
	Mat image = imread("C:\\Users\\Adam\\Desktop\\samples\\completed.jpg", 1);
	cols = image.cols;
	rows = image.rows;

	step = (sqrt((cols * rows) / (double)numberofSuperpixels));

	initData(image);
	dataCopy();

	int threadsToBeStarted = rows*cols + centersLength - 1;
	int howManyBlocks = threadsToBeStarted / 700;
	int threadsPerBlock = (threadsToBeStarted / howManyBlocks) + 1;

	//for (int i = 0; i < iteration; i++)
	//{
		compute << <howManyBlocks, threadsPerBlock >> > (cols, rows, step, centersLength, d_clusters, d_distances, d_centers, d_center_counts, d_colors, 5);
		compute1 << <howManyBlocks, threadsPerBlock >> > (cols, rows, step, centersLength, d_clusters, d_distances, d_centers, d_center_counts, d_colors, 5);
	//}

	hipMemcpy(distances, d_distances, sizeof(float)*rows*cols, hipMemcpyDeviceToHost);
	hipMemcpy(clusters, d_clusters, sizeof(float)*rows*cols, hipMemcpyDeviceToHost);
	hipMemcpy(centers, d_centers, sizeof(float)*centersLength * 5, hipMemcpyDeviceToHost);
	hipMemcpy(center_counts, d_center_counts, sizeof(float)*centersLength, hipMemcpyDeviceToHost);

	dataFree();

	int a = 0;
	for (int i = 0; i < rows*cols; i++)
	{
		if (clusters[i] == -1)
		{
			a++;
		}
	}
	int b = rows*cols - a;

	printf("%i steps\n", step);
	printf("%i rows\n", rows);
	printf("%i cols\n", cols);
	printf("%i darab cluster\n", centersLength);
	printf("%i darab pixel\n", rows*cols);
	printf("%i darab elinditott szal\n", threadsPerBlock*howManyBlocks);
	printf("%i darab clusterhez van renderve\n", b);
	printf("%i darab nincs clusterhez renderve\n", a);
	
	int dis = 0;
	for (int i = 0; i < rows*cols; i++)
	{
		if (distances[i] == FLT_MAX)
		{
			dis++;
		}
	}
	printf("%i dis\n", dis);


	int mennyi = 0;
	for (int i = 0; i < centersLength; i++)
	{
		//cout << center_counts[i] << endl;
		mennyi += center_counts[i];
	}
	printf("%i mennyi\n", mennyi);

	//getchar();

	//getchar();
	//for (int i = 0; i < rows*cols; i++)
	//{
	//	cout << distances[i] << endl;
	//}

	//for (int i = 0; i < centersLength; i += 5)
	//{
	//	cout << centers[i] << " " << centers[i + 1] << " " << centers[i + 2] << " " << centers[i + 3] << " " << centers[i + 4] << endl;
	//}

	printf("vege");
	///* Load the image and convert to Lab colour space. */
	//Mat image = imread("C:\\Users\\Adam\\Desktop\\samples\\completed.jpg", 1);
	//Mat lab_image = image.clone();
	//cvtColor(image, lab_image, CV_BGR2Lab);

	///* Yield the number of superpixels and weight-factors from the user. */
	//int w = image.cols;
	//int h = image.rows;
	//int nr_superpixels = 5000;
	//int nc = 80;

	//double step = (sqrt((w * h) / (double)nr_superpixels));
	////1400*900-as k�pn�l, 1000 superpixellel --> 35,496 --> v�zszintesen 39,444, f�gg�legesen 25,354

	///* Perform the SLIC superpixel algorithm. */
	//Slic slic;
	//slic.generate_superpixels(lab_image, step, nc);
	//slic.create_connectivity(lab_image);

	///* Display the contours and show the result. */
	//Mat tt = image.clone();
	//slic.display_contours(tt, Vec3b(0, 0, 255));
	//imwrite("C:\\Users\\Adam\\Desktop\\0MATsamplewitchLines.jpg", tt);

	////----------------------
	//slic.colour_with_cluster_means(image);
	//imwrite("C:\\Users\\Adam\\Desktop\\1MATsamplefilled.jpg", image);
	////----------------------

	getchar();
}
