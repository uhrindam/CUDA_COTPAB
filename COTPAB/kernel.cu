#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2\opencv.hpp>
//#include <opencv2\cudaimgproc.hpp>

#include "slic.h"

using namespace std;
using namespace cv;

#define nc 80 //maximum vizsg�lt t�vols�g a centroidok keres�sekor
#define numberofSuperpixels 5000

int cols;
int rows;
int step;
int centersLength;
int *clusters;
float *distances;
float *centers;
int *center_counts;
uchar3 *colors;

__device__ int d_cols;
__device__ int d_rows;
__device__ int d_step;
__device__ int d_centersLength;
__device__ int *d_clusters;			//1D --> cols * rows
__device__ float *d_distances;		//1D --> cols * rows
__device__ float *d_centers;		//2D --> centersLength * 5
__device__ int *d_center_counts;	//1D --> centersLength
__device__ uchar3 *d_colors;		//1D --> cols * rows

__device__ float compute_dist(int ci, int x, int y, uchar3 colour, float *d_centers, int pitch, int d_step)
{
	//sz�nt�vols�g
	float dc = sqrt(pow(d_centers[ci *pitch  + 0] - colour.x, 2) + pow(d_centers[ci *pitch  + 1] - colour.y, 2)
		+ pow(d_centers[ci *pitch  + 2] - colour.z, 2));
	//euklideszi t�vols�g
	float ds = sqrt(pow(d_centers[ci *pitch  + 3] - x, 2) + pow(d_centers[ci *pitch  + 4] - y, 2));

	return sqrt(pow(dc / nc, 2) + pow(ds / d_step, 2));
}

__global__ void compute(int d_cols, int d_rows, int d_step, int d_centersLength, int *d_clusters,  float *d_distances, 
	float *d_centers, int *d_center_counts, uchar3 *d_colors, int pitch)
{
	d_distances[threadIdx.x] = compute_dist(885, threadIdx.x % d_rows, threadIdx.x / d_rows, d_colors[threadIdx.x], d_centers, pitch, d_step);
}

void initData(Mat image)
{
	clusters = new int[cols*rows];
	distances = new float[cols*rows];
	for (int i = 0; i < cols*rows; i++)
	{
		clusters[i] = -1;
		distances[i] = FLT_MAX;
	}

	//Ez az�rt kell mert el�re nem tudom, hogy h�ny eleme lesz a centers-nek, ez�rt el�sz�r egy vectorhoz adomgatom hozz� az elemeket
	// majd k�s�bb l�trehozom a t�mb�t annyi elemmel, ah�ny eleme van a seg�dvectornak, majd �tm�solom az adatokat.
	vector<vector<float> > h_centers;
	for (int i = step; i < cols - step / 2; i += step) {
		for (int j = step; j < rows - step / 2; j += step) {
			vector<float> center;
			/* Find the local minimum (gradient-wise). */
			//Point nc = find_local_minimum(image, Point(i, j));
			Vec3b colour = image.at<Vec3b>(j, i);//nc.y, nc.x);

			center.push_back(colour.val[0]);
			center.push_back(colour.val[1]);
			center.push_back(colour.val[2]);
			center.push_back(i);//nc.x);
			center.push_back(j);//nc.y);

			h_centers.push_back(center);
		}
	}

	centersLength = h_centers.size();

	centers = new float[centersLength * 5];
	center_counts = new int[centersLength];
	int idx = 0;
	for (int i = 0; i < centersLength; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			centers[idx] = h_centers[i][j];
			idx++;
		}

		center_counts[i] = 0;
	}

	//Bej�rom a k�pet, majd minden pixel sz�n�t (3 �rt�k) elmentem egy uchar3 v�ltoz�ba
	colors = new uchar3[rows*cols];
	for (int i = 0; i < cols; i++)
	{
		for (int j = 0; j < rows; j++)
		{
			Vec3b colour = image.at<Vec3b>(j, i);
			colors[i * rows + j] = make_uchar3(colour.val[0], colour.val[1], colour.val[2]);
		}
	}
}

void dataCopy()
{
	hipMalloc((void**)&d_cols, sizeof(int));
	hipMemcpy(&d_cols, &cols, sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_rows, sizeof(int));
	hipMemcpy(&d_rows, &rows, sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_step, sizeof(int));
	hipMemcpy(&d_step, &step, sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_centersLength, sizeof(int));
	hipMemcpy(&d_centersLength, &centersLength, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_clusters, sizeof(int)*rows*cols);
	hipMemcpy(d_clusters, clusters, sizeof(int)*rows*cols, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_distances, sizeof(float)*rows*cols);
	hipMemcpy(d_distances, distances, sizeof(float)*rows*cols, hipMemcpyHostToDevice);

	size_t pitch = 5;
	hipMallocPitch((void**)&d_centers, &pitch, sizeof(float) * centersLength, 5);
	hipMemcpy2D(d_centers, pitch, centers, sizeof(float) * centersLength, sizeof(float) * centersLength, 5, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_center_counts, sizeof(int)*centersLength);
	hipMemcpy(d_center_counts, center_counts, sizeof(int)*centersLength, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_colors, sizeof(uchar3)*rows*cols);
	hipMemcpy(d_colors, colors, sizeof(uchar3)*rows*cols, hipMemcpyHostToDevice);
}

void dataFree()
{
	hipFree(d_clusters);
	hipFree(d_distances);
	hipFree(d_centers);
	hipFree(d_center_counts);
	hipFree(d_colors);
}

int main()
{
	Mat image = imread("C:\\Users\\Adam\\Desktop\\samples\\completed.jpg", 1);
	cols = image.cols;
	rows = image.rows;

	step = (sqrt((cols * rows) / (double)numberofSuperpixels));

	initData(image);
	dataCopy();

	int pitchInt = 5;
	compute << <1, 100 >> > (d_cols, d_rows, d_step, d_centersLength, d_clusters, d_distances, d_centers, d_center_counts, d_colors, pitchInt);

	hipMemcpy(distances, d_distances, sizeof(float)*rows*cols, hipMemcpyDeviceToHost);
	
	dataFree();

	for (int i = 0; i < 101; i++)
	{
		cout << distances[i] << endl;
	}

	printf("vege");
	///* Load the image and convert to Lab colour space. */
	//Mat image = imread("C:\\Users\\Adam\\Desktop\\samples\\completed.jpg", 1);
	//Mat lab_image = image.clone();
	//cvtColor(image, lab_image, CV_BGR2Lab);

	///* Yield the number of superpixels and weight-factors from the user. */
	//int w = image.cols;
	//int h = image.rows;
	//int nr_superpixels = 5000;
	//int nc = 80;

	//double step = (sqrt((w * h) / (double)nr_superpixels));
	////1400*900-as k�pn�l, 1000 superpixellel --> 35,496 --> v�zszintesen 39,444, f�gg�legesen 25,354

	///* Perform the SLIC superpixel algorithm. */
	//Slic slic;
	//slic.generate_superpixels(lab_image, step, nc);
	//slic.create_connectivity(lab_image);

	///* Display the contours and show the result. */
	//Mat tt = image.clone();
	//slic.display_contours(tt, Vec3b(0, 0, 255));
	//imwrite("C:\\Users\\Adam\\Desktop\\0MATsamplewitchLines.jpg", tt);

	////----------------------
	//slic.colour_with_cluster_means(image);
	//imwrite("C:\\Users\\Adam\\Desktop\\1MATsamplefilled.jpg", image);
	////----------------------

	getchar();
}
